#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include <map>

#include "utils.hpp"
#include "Graph/graph.hpp"
#include "Cuda/CudaFunctions.cuh"

#define ll long long
#define DEFAULT_SEED 1234
#define DEFAULT_BLOCK_SIZE 16

/*
    -s: seed
    -c: check / verify results
    -V: (verbose) print results matrix
    -v: vectorized if possible with short4
    -b <block size>: Set block size for GPU
    -p <percentage>: Set percentage for Erdos-Renyi graph generation
    -a <algorithm>: Set algorithm to use (1: cpu, 1: simple, 2: blocked)
*/
int main(int argc, char **argv){
    int perc = 50, blockSize = DEFAULT_BLOCK_SIZE, algorithm = 0, seed = DEFAULT_SEED;
    bool toVerify = false, printResults = false, vectorize = false;

    if(argc < 2 || argc > 13)
        throw std::invalid_argument("Utilizzo comando: ./parallel_fw [-s] seed [-p] percentage [-b] BlockSize [-a] algorithm [-c] [-V] [-v] numVertices");
        
    short *graph = nullptr;
    const ll numVertices = atoll(argv[argc - 1]);

    int opt;
    extern char *optarg;
    std::map<short, short> sqrts = {{1024, 32}, {256, 16}, {64, 8}, {16, 4}};
    while((opt = getopt(argc, argv, "s:p:b:a:cvV")) != -1){
        switch(opt){
            case 's':
                seed = atoi(optarg);
                break;
            case 'p':
                perc = atoi(optarg);
                if(perc <= 0 || perc >= 100)
                    throw std::invalid_argument("Inserire percentuale compreso tra 0 e 100 (estremi esclusi)");
                break;
            case 'b':
                blockSize = atoi(optarg);
                if(sqrts.find(blockSize) == sqrts.end())
                    throw std::invalid_argument("Invalid block size: 1024, 256, 64, 16");

                blockSize = sqrts[blockSize];
                break;
            case 'a':
                algorithm = atoi(optarg);
                if(algorithm == 0 || (algorithm != 1 && algorithm != 2 && algorithm != 3))
                    throw std::invalid_argument("Inserire 1 per FW su CPU, 2 per FW parallelizzato su global memory, 3 per FW parallelizzato su shared memory (blocked)");
                break;
            case 'c':
                toVerify = true;
                break;
            case 'v':
                vectorize = true;
                break;
            case 'V':
                printResults = true;
                break;
            default:
                throw std::invalid_argument("Utilizzo comando: ./parallel_fw [-s] seed [-p] percentage [-b] BlockSize [-a] algorithm [-c] [-V] [-v] numVertices");
        }
    }

    if(vectorize && (numVertices & 3))
        throw std::invalid_argument("Il numero di vertici deve essere multiplo di 4 per poter utilizzare la versione vectorized");

    ll numCol = numVertices;
    if(algorithm == 3){
        const int remainder = numVertices - blockSize * (numVertices / blockSize);
        if (remainder)
            numCol = numVertices + blockSize - remainder;

        graph = blockedGraphInit(numVertices, perc, blockSize, seed);
    }
    else
        graph = graphInit(numVertices, perc, seed);

    //! ------------ PARALLEL FLOYD WARSHALL ON GPU -----

    short *w_GPU = nullptr;
    switch (algorithm){
        case 1:
            w_GPU = FloydWarshallCPU(graph, numVertices, numCol);
            break;
        case 2:
            w_GPU = simple_parallel_FW(graph, numCol, blockSize, vectorize);
            break;
        case 3:
            w_GPU = blocked_parallel_FW(graph, numCol, blockSize, vectorize);
            break;
    }

    //! ----------------------------------------------

    //! ------------------ VERIFY --------------------
    
    if(toVerify){
        short *resultsForVerify = FloydWarshallCPU(graph, numVertices, numCol);
        
        verify(resultsForVerify, numVertices, w_GPU, numCol);
        delete[] resultsForVerify;
    }

    if (printResults){
        std::cout << "Originale: " << std::endl;
        printMatrix(graph, numVertices, numCol);
        std::cout << "Risultato: " << std::endl;
        printMatrix(w_GPU, numVertices, numCol);
    }

    //! -----------------------------------------------------------

    if(algorithm == 1)
        delete[] w_GPU;
    else
        cuda(hipHostFree(w_GPU));

    delete[] graph;
    exit(0);
}
