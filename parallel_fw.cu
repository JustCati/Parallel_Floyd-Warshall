#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include <map>

#include "utils.hpp"
#include "Graph/graph.hpp"
#include "Cuda/CudaFunctions.cuh"

#define CPU_VERT_LIMIT 4096
#define DEFAULT_BLOCK_SIZE 16
#define ll long long

/*
    -P: use pitch
    -c: check / verify results
    -V: (verbose) print results matrix
    -v: vectorized if possible with short4
    -b <block size>: Set block size for GPU
    -p <percentage>: Set percentage for Erdos-Renyi graph generation
    -a <algorithm>: Set algorithm to use (1: cpu, 1: simple, 2: blocked)
*/
int main(int argc, char **argv){
    bool usePitch = false, vectorize = false;
    bool toVerify = false, printResults = false;
    int perc = 50, blockSize = DEFAULT_BLOCK_SIZE, algorithm = 0;

    if(argc < 2 || argc > 12)
        throw std::invalid_argument("Utilizzo comando: ./parallel_fw num_vertices [-p] percentage [-b] BlockSize [-a] algorithm [-c] [-V] [-v] [-P]");
        
    short* graph = nullptr;
    const ll numVertices = atoll(argv[argc - 1]);

    int opt;
    extern char *optarg;
    std::map<short, short> sqrts = {{1024, 32}, {256, 16}, {64, 8}, {16, 4}};
    while((opt = getopt(argc, argv, "p:b:a:cvVP")) != -1){
        switch(opt){
            case 'p':
                perc = atoi(optarg);
                if(perc <= 0 || perc >= 100)
                    throw std::invalid_argument("Inserire percentuale compreso tra 0 e 100 (estremi esclusi)");
                break;
            case 'b':
                blockSize = atoi(optarg);
                if(sqrts.find(blockSize) == sqrts.end())
                    throw std::invalid_argument("Invalid block size for blocked parallel FW algorithm");

                blockSize = sqrts[blockSize];
                break;
            case 'a':
                algorithm = atoi(optarg);
                if(algorithm == 0 || (algorithm != 1 && algorithm != 2 && algorithm != 3))
                    throw std::invalid_argument("Inserire 1 per FW su CPU, 2 per FW parallelizzato su global memory, 3 per FW parallelizzato su shared memory (blocked)");
                break;
            case 'c':
                toVerify = true;
                break;
            case 'P':
                usePitch = true;
                break;
            case 'v':
                vectorize = true;
                break;
            case 'V':
                printResults = true;
                break;
            default:
                throw std::invalid_argument("Utilizzo comando: ./parallel_fw num_vertices [-p] percentage [-b] BlockSize [-a] algorithm [-c] [-V] [-v] [-P]");
        }
    }

    if(vectorize && (numVertices & 3))
        throw std::invalid_argument("Il numero di vertici deve essere multiplo di 4 per poter utilizzare la versione vectorized");

    ll numCol = numVertices;
    if(algorithm == 3){
        const int remainder = numVertices - blockSize * (numVertices / blockSize);
        if (remainder)
            numCol = numVertices + blockSize - remainder;

        graph = blockedGraphInit(numVertices, perc, blockSize);
    }
    else
        graph = graphInit(numVertices, perc);

    //! ------------ PARALLEL FLOYD WARSHALL ON GPU -----

    short* w_GPU = nullptr;
    switch (algorithm){
        case 1:
            w_GPU = FloydWarshallCPU(graph, numVertices, numCol);
            break;
        case 2:
            w_GPU = simple_parallel_FW(graph, numCol, blockSize, usePitch, vectorize);
            break;
        case 3:
            w_GPU = blocked_parallel_FW(graph, numCol, blockSize, usePitch, vectorize);
            break;
    }

    //! ----------------------------------------------

    //! ------------------ VERIFY --------------------
    if(toVerify){
        bool cpuExec = true;
        short *resultsForVerify = nullptr;
        
        // If the number of vertices is too high, the CPU version will be too slow
        if (numVertices < CPU_VERT_LIMIT)
            resultsForVerify = FloydWarshallCPU(graph, numVertices, numCol);
        else{
            cpuExec = false;
            resultsForVerify = simple_parallel_FW(graph, numCol, DEFAULT_BLOCK_SIZE, false, false, true);
        }

        verify(resultsForVerify, numVertices, w_GPU, numCol);

        if(cpuExec)
            delete[] resultsForVerify;
        else
            cuda(hipHostFree(resultsForVerify));
    }

    if (printResults)
        printMatrix(w_GPU, numVertices, numCol);

    //! -----------------------------------------------------------

    if(algorithm == 1)
        delete[] w_GPU;
    else
        cuda(hipHostFree(w_GPU));

    delete[] graph;
    exit(0);
}
