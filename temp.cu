#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>


__global__ void kernel(short *A, size_t pitch, int n){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    short *arr = (short*)((char*)A + i * pitch);
    if(i >= n || j >= pitch)
        arr[j] = 5;
    else
        arr[j] = 1;
}


int main(){
    const int n = 4;
    size_t pitch;
    short *A;

    hipMallocPitch(&A, &pitch, n * sizeof(short), n);

    std::cout << "pitch: " << pitch  << " widht: "<< 10*sizeof(short) <<  std::endl;

    dim3 block(16, 16);
    dim3 grid(pitch, n);
    kernel<<<grid, block>>>(A, pitch, n);

    hipDeviceSynchronize();

    short *h_A;
    hipHostMalloc(&h_A, pitch * 10, hipHostMallocDefault);
    hipMemcpy2D(h_A, pitch, A, pitch, pitch, n, hipMemcpyDeviceToHost);
    
    std::ofstream out("out.txt");
    for(int i = 0; i < n; i++){
        for(int j = 0; j < pitch; j++)
            out << h_A[i * pitch + j] << "\t";
        out << std::endl;
    }
    out.close();
    
    hipHostFree(h_A);
    hipFree(A);
    return 0;
}

