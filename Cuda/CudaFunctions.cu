#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include "CudaFunctions.cuh"
#include "../Graph/graph.hpp"





int* simple_parallel_FW(const Graph& g){
    size_t memsize = g.getMatrixSize();
    const int* matrix = g.getAdjMatrix();

    int* d_matrix;
    cuda(hipMalloc(&d_matrix, memsize));
    cuda(hipMemcpy(d_matrix, matrix, memsize, hipMemcpyHostToDevice));

    //* ----- INIT AND CALL KERNEL ------
    dim3 dimBlock = dim3(32, 32);
    dim3 numBlock = dim3((g.getNumVertices() + dimBlock.x - 1) / dimBlock.x, (g.getNumVertices() + dimBlock.x - 1) / dimBlock.y);

    for(int k = 0; k < g.getNumVertices(); k++)
        FW_simple_kernel<<<numBlock, dimBlock>>>(d_matrix, g.getNumVertices(), k);
    //* ----------------------------------

    int* h_matrix;
    cuda(hipHostMalloc(&h_matrix, memsize));
    cuda(hipMemcpy(h_matrix, d_matrix, memsize, hipMemcpyDeviceToHost));

    cuda(hipFree(d_matrix));
    return h_matrix;
}
