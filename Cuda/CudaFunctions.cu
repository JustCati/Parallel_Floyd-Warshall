#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include "CudaFunctions.cuh"
#include "../Graph/graph.hpp"


int* simple_parallel_FW(const Graph& g){
    int* d_matrix;
    size_t memsize = g.getMatrixMemSize();
    const int* matrix = g.getAdjMatrix();


    float elapsedTime;
    hipEvent_t start, stop;
    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));


    cuda(hipEventRecord(start));
    cuda(hipMalloc(&d_matrix, memsize));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMalloc time: " << elapsedTime << " ms" << std::endl;

    cuda(hipEventRecord(start));
    cuda(hipMemcpy(d_matrix, matrix, memsize, hipMemcpyHostToDevice));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));    
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMemCpy to device time: " << elapsedTime << " ms" << std::endl;


    
    //* ----- INIT AND CALL KERNEL ------
    const int blockSize = g.getBlockSize();
    dim3 dimBlock = dim3(blockSize, blockSize);
    dim3 numBlock = dim3((g.getNumVertices() + dimBlock.x - 1) / dimBlock.x, (g.getNumVertices() + dimBlock.x - 1) / dimBlock.y);

    cuda(hipEventRecord(start));

    for(int k = 0; k < g.getNumVertices(); k++)
        FW_simple_kernel<<<numBlock, dimBlock>>>(d_matrix, g.getNumVertices(), k);
        
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    //* ----------------------------------

    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "Total kernel time: " << elapsedTime << " ms" << std::endl;

    int* h_matrix;
    cuda(hipEventRecord(start));
    cuda(hipHostMalloc(&h_matrix, memsize));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMallocHost time: " << elapsedTime << " ms" << std::endl;

    cuda(hipEventRecord(start));
    cuda(hipMemcpy(h_matrix, d_matrix, memsize, hipMemcpyDeviceToHost));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMemCpy to host time: " << elapsedTime << " ms" << std::endl;

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}
