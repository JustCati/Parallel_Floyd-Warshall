#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <numeric>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include "CudaFunctions.cuh"
#include "../Graph/graph.hpp"

#define DEFAULT_BLOCK_SIZE 32



void printMetrics(std::string title, std::vector<std::string> outputs, std::vector<float> times){
    if (outputs.size() != times.size()){
        std::cout << "ERROR: outputs and times vectors are not the same size" << std::endl;
        return;
    }
    std::cout << title << std::endl << std::endl;
    for(int i = 0; i < outputs.size(); i++){
        std::cout << outputs[i] << times[i];
        if(outputs[i].find("Bandwidth") == std::string::npos)
            std::cout << " ms";
        else
            std::cout << " GB/s";
        std::cout << std::endl;
    }
    
    std::cout << std::endl;
    std::cout << "Total Kernel time: " << std::accumulate(times.begin(), times.end(), 0.0) / 1000 << " s" << std::endl;
}

short* simple_parallel_FW(const short* g, int numVertices, int blockSize, bool usePitch, bool vectorize, bool debug){
    size_t pitch = 0;
    short* d_matrix, *h_matrix;
    size_t singleRow_memsize, memsize;

    if (vectorize){
        singleRow_memsize = (numVertices >> 2) * sizeof(short4);
        memsize = (numVertices >> 2) * numVertices * sizeof(short4);
    }
    else{
        singleRow_memsize = numVertices * sizeof(short);
        memsize = numVertices * numVertices * sizeof(short);
    }

    float elapsedTime;
    std::vector<float> times;
    std::vector<std::string> outputs;

    hipEvent_t start, stop;
    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));

    cuda(hipEventRecord(start));
    if (usePitch){
        cuda(hipMallocPitch(&d_matrix, &pitch, singleRow_memsize, numVertices)); //* allocate memory on device
    }
    else
        cuda(hipMalloc(&d_matrix, memsize)); //* allocate memory on device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMalloc: ");
    times.push_back(elapsedTime);

    cuda(hipEventRecord(start));
    if (usePitch){
        cuda(hipMemcpy2D(d_matrix, pitch, g, singleRow_memsize, singleRow_memsize, numVertices, hipMemcpyHostToDevice)); //* copy matrix to device
    }
    else
        cuda(hipMemcpy(d_matrix, g, memsize, hipMemcpyHostToDevice)); //* copy matrix to device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to device: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to device Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    cuda(hipEventRecord(start));

    //* ---------------------- KERNEL ---------------------- *//

    if(!vectorize){
        dim3 dimBlock = dim3(blockSize, blockSize);
        dim3 numBlock = dim3((numVertices + dimBlock.x - 1) / dimBlock.x, (numVertices + dimBlock.y - 1) / dimBlock.y);

        if(usePitch)
            for(int k = 0; k < numVertices; k++)
                FW_simple_kernel_pitch<<<numBlock, dimBlock>>>(d_matrix, pitch, numVertices, k); //* call kernel
        else
            for(int k = 0; k < numVertices; k++)
                FW_simple_kernel<<<numBlock, dimBlock>>>(d_matrix, numVertices, k); //* call kernel
    }
    else{ //* vectorize with short4 type (default)
        dim3 dimBlock = dim3(blockSize, blockSize);
        dim3 numBlock = dim3((numVertices + dimBlock.x - 1) / dimBlock.x, (numVertices + dimBlock.y - 1) / dimBlock.y);

        if(usePitch)
            for(int k = 0; k < numVertices; k++)
                FW_simple_kernel_vectorized_pitch<<<numBlock, dimBlock>>>((short4*)d_matrix, pitch, numVertices, k); //* call kernel
        else
            for(int k = 0; k < numVertices; k++)
                FW_simple_kernel_vectorized<<<numBlock, dimBlock>>>((short4*)d_matrix, numVertices, k); //* call kernel
    }
    
    //* ---------------------------------------------------- *//
    
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("Total kernel call: ");
    times.push_back(elapsedTime);

    cuda(hipEventRecord(start));
    cuda(hipHostMalloc(&h_matrix, memsize)); //* allocate memory on host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMallocHost: ");
    times.push_back(elapsedTime);

    cuda(hipEventRecord(start));
    if (usePitch){
        cuda(hipMemcpy2D(h_matrix, singleRow_memsize, d_matrix, pitch, singleRow_memsize, numVertices, hipMemcpyDeviceToHost)); //* copy matrix to host
    }
    else
        cuda(hipMemcpy(h_matrix, d_matrix, memsize, hipMemcpyDeviceToHost)); //* copy matrix to host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to Host: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to Host Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    if(!debug){
        std::string title =  "Starting SIMPLE FW KERNEL with " + std::to_string(numVertices) +\
        " nodes" + (usePitch ? " with pitch, " : "") + (vectorize ? " with vectorization" : "");
        printMetrics(title, outputs, times); //* print metrics
    }

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}

short* blocked_parallel_FW(const short* g, int numVertices, int blockSize){
    short* d_matrix, *h_matrix;
    size_t memsize = numVertices * numVertices * sizeof(short);

    float elapsedTime;
    std::vector<float> times;
    std::vector<std::string> outputs;

    hipEvent_t start, stop;
    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));

    cuda(hipEventRecord(start));
    cuda(hipMalloc(&d_matrix, memsize)); //* allocate memory on device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMalloc: ");
    times.push_back(elapsedTime);

    cuda(hipEventRecord(start));
    cuda(hipMemcpy(d_matrix, g, memsize, hipMemcpyHostToDevice)); //* copy matrix to device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to device: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to device Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    cuda(hipEventRecord(start));

    //* ---------------------- KERNEL ---------------------- *//
    const int numBlocks = (numVertices + blockSize - 1) / blockSize;
    dim3 dimBlock = dim3(blockSize, blockSize);
    dim3 dimBlock_phase3 = dim3(numBlocks, numBlocks);
    size_t sharedMemSize = blockSize * blockSize * sizeof(int);

    for(int k = 0; k < numBlocks; k++){
        blocked_FW_phase1<<<1, dimBlock, sharedMemSize>>>(d_matrix, numVertices, k, blockSize);
        blocked_FW_phase2<<<numBlocks, dimBlock, 2 * sharedMemSize>>>(d_matrix, numVertices, k, blockSize);
        blocked_FW_phase3<<<dimBlock_phase3, dimBlock, 3 * sharedMemSize>>>(d_matrix, numVertices, k, blockSize);
    }
    //* ------------------------------------------------------ *//
    
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("Total kernel call: ");
    times.push_back(elapsedTime);

    cuda(hipEventRecord(start));
    cuda(hipHostMalloc(&h_matrix, memsize)); //* allocate memory on host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMallocHost: ");
    times.push_back(elapsedTime);

    cuda(hipEventRecord(start));
    cuda(hipMemcpy(h_matrix, d_matrix, memsize, hipMemcpyDeviceToHost)); //* copy matrix to host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to host: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to host Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    std::string title =  "Starting BLOCKED FW KERNEL with " + std::to_string(numVertices) + " nodes";
    printMetrics(title, outputs, times); //* print metrics

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}
