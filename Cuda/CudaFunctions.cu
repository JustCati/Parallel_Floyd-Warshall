#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include "CudaFunctions.cuh"
#include "../Graph/graph.hpp"


int* simple_parallel_FW(const Graph& g){
    size_t memsize = g.getMatrixSize();
    const int* matrix = g.getAdjMatrix();

    int* d_matrix;
    float elapsedTime;
    hipEvent_t start, stop;

    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));


    cuda(hipEventRecord(start));
    cuda(hipMalloc(&d_matrix, memsize));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMalloc time: " << elapsedTime << " ms" << std::endl;

    cuda(hipEventRecord(start));
    cuda(hipMemcpy(d_matrix, matrix, memsize, hipMemcpyHostToDevice));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));    
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMemCpy to device time: " << elapsedTime << " ms" << std::endl;


    float totalElapsedTime = 0;
    //* ----- INIT AND CALL KERNEL ------
    dim3 dimBlock = dim3(32, 32);
    dim3 numBlock = dim3((g.getNumVertices() + dimBlock.x - 1) / dimBlock.x, (g.getNumVertices() + dimBlock.x - 1) / dimBlock.y);
    
    for(int k = 0; k < g.getNumVertices(); k++){
        cuda(hipEventRecord(start));
        FW_simple_kernel<<<numBlock, dimBlock>>>(d_matrix, g.getNumVertices(), k);
        cuda(hipEventRecord(stop));
        cuda(hipEventSynchronize(stop));
        cuda(hipEventElapsedTime(&elapsedTime, start, stop));

        totalElapsedTime += elapsedTime;
    }
    //* ----------------------------------

    std::cout << "Total kernel time: " << totalElapsedTime << " ms" << std::endl;

    int* h_matrix;
    cuda(hipEventRecord(start));
    cuda(hipHostMalloc(&h_matrix, memsize));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMallocHost time: " << elapsedTime << " ms" << std::endl;

    cuda(hipEventRecord(start));
    cuda(hipMemcpy(h_matrix, d_matrix, memsize, hipMemcpyDeviceToHost));
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "CudaMemCpy to host time: " << elapsedTime << " ms" << std::endl;

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}
