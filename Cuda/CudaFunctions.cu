#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <numeric>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include "CudaFunctions.cuh"
#include "../Graph/graph.hpp"

#define ll long long


void printMetrics(std::string title, std::vector<std::string> outputs, std::vector<float> times){
    if (outputs.size() != times.size())
        throw std::runtime_error("Outputs e times vectors sono di dimensioni diverse");
    
    std::cout << title << std::endl << std::endl;
    for(int i = 0; i < outputs.size(); i++){
        std::cout << outputs[i] << times[i];
        if(outputs[i].find("Bandwidth") == std::string::npos)
            std::cout << " ms";
        else
            std::cout << " GB/s";
        std::cout << std::endl;
    }
    
    std::cout << std::endl;
    std::cout << "Total Function time: " << std::accumulate(times.begin(), times.end(), 0.0) / 1000 << " s" << std::endl;
}

short* simple_parallel_FW(const short *g, ll numVertices, int blockSize, bool vectorize, bool debug){
    size_t pitch = 0;
    short *d_matrix, *h_matrix;
    const size_t singleRow_memsize = numVertices * sizeof(short);
    const size_t memsize = numVertices * numVertices * sizeof(short);

    
    float elapsedTime;
    std::vector<float> times;
    std::vector<std::string> outputs;

    hipEvent_t start, stop;
    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));

    cuda(hipMallocPitch(&d_matrix, &pitch, singleRow_memsize, numVertices)); //* allocate memory on device

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(d_matrix, pitch, g, singleRow_memsize, singleRow_memsize, numVertices, hipMemcpyHostToDevice)); //* copy matrix to device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to device: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to device Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    cuda(hipEventRecord(start));

    //* ---------------------- KERNEL ---------------------- *//
    dim3 dimBlock = dim3(blockSize, blockSize);
    dim3 numBlock = dim3((numVertices + dimBlock.x - 1) / dimBlock.x, (numVertices + dimBlock.y - 1) / dimBlock.y);

    if(vectorize){ //* vectorize with short4 type (default)
        dimBlock = dim3(blockSize >> 2, blockSize);
        for(int k = 0; k < numVertices; k++)
            FW_simple_kernel_vectorized<<<numBlock, dimBlock>>>((short4*)d_matrix, pitch, numVertices >> 2, k); //* call kernel
    }
    else
        for(int k = 0; k < numVertices; k++)
            FW_simple_kernel<<<numBlock, dimBlock>>>(d_matrix, pitch, numVertices, k); //* call kernel

    //* ---------------------------------------------------- *//

    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("Total kernel call: ");
    times.push_back(elapsedTime);

    cuda(hipHostMalloc(&h_matrix, memsize)); //* allocate memory on host

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(h_matrix, singleRow_memsize, d_matrix, pitch, singleRow_memsize, numVertices, hipMemcpyDeviceToHost)); //* copy matrix to host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to Host: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to Host Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    if(!debug){
        std::string title =  "Starting SIMPLE FW KERNEL with " + std::to_string(numVertices) +\
        " nodes" + (vectorize ? " with vectorization" : "");
        printMetrics(title, outputs, times); //* print metrics
    }

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}


short* blocked_parallel_FW(const short *g, ll numVertices, int blockSize, bool vectorize){
    size_t pitch = 0;
    short *d_matrix, *h_matrix;
    const size_t singleRow_memsize = numVertices * sizeof(short);
    const size_t memsize = numVertices * numVertices * sizeof(short);


    float elapsedTime;
    std::vector<float> times;
    std::vector<std::string> outputs;

    hipEvent_t start, stop;
    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));

    cuda(hipMallocPitch(&d_matrix, &pitch, singleRow_memsize, numVertices)); //* allocate memory on device

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(d_matrix, pitch, g, singleRow_memsize, singleRow_memsize, numVertices, hipMemcpyHostToDevice)); //* copy matrix to device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to device: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to device Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    cuda(hipEventRecord(start));

    //* ---------------------- KERNEL ---------------------- *//
    const int numBlocks = (numVertices + blockSize - 1) / blockSize;

    dim3 dimBlock = dim3(blockSize, blockSize);
    dim3 dimBlock_phase3 = dim3(numBlocks, numBlocks);
    const size_t sharedMemSize = blockSize * blockSize * sizeof(short);

    if(vectorize){
        dimBlock = dim3(blockSize >> 2, blockSize);

        for(int k = 0; k < numBlocks; k++){
            blocked_FW_phase1_vectorized<<<1, dimBlock, sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase2_vectorized<<<numBlocks, dimBlock, 2 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase3_vectorized<<<dimBlock_phase3, dimBlock, 3 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
        }
    }
    else{
        for(int k = 0; k < numBlocks; k++){
            blocked_FW_phase1<<<1, dimBlock, sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase2<<<numBlocks, dimBlock, 2 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase3<<<dimBlock_phase3, dimBlock, 3 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
        }
    }
    //* ------------------------------------------------------ *//

    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("Total kernel call: ");
    times.push_back(elapsedTime);

    cuda(hipHostMalloc(&h_matrix, memsize)); //* allocate memory on host

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(h_matrix, singleRow_memsize, d_matrix, pitch, singleRow_memsize, numVertices, hipMemcpyDeviceToHost)); //* copy matrix to host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to host: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to host Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    std::string title =  "Starting BLOCKED FW KERNEL with " + std::to_string(numVertices) +\
    " nodes" + (vectorize ? " with vectorization" : "");
    printMetrics(title, outputs, times); //* print metrics

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}
