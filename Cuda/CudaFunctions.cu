#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <numeric>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include "CudaFunctions.cuh"
#include "../Graph/graph.hpp"

#define ll long long


void printMetrics(std::string title, std::vector<std::string> outputs, std::vector<float> times){
    if (outputs.size() != times.size())
        throw std::runtime_error("Outputs e times vectors sono di dimensioni diverse");
    
    std::cout << title << std::endl << std::endl;
    for(int i = 0; i < outputs.size(); i++){
        std::cout << outputs[i] << times[i];
        if(outputs[i].find("Bandwidth") == std::string::npos)
            std::cout << " ms";
        else
            std::cout << " GB/s";
        std::cout << std::endl;
    }
}


short* simple_parallel_FW(const short *g, ll numVertices, int blockSize, bool vectorize){
    size_t pitch = 0;
    short *d_matrix, *h_matrix;
    const size_t singleRow_memsize = numVertices * sizeof(short);
    const size_t memsize = numVertices * numVertices * sizeof(short);

    
    float elapsedTime;
    std::vector<float> times;
    std::vector<std::string> outputs;

    hipEvent_t start, stop;
    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));

    cuda(hipMallocPitch(&d_matrix, &pitch, singleRow_memsize, numVertices)); //* allocate memory on device

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(d_matrix, pitch, g, singleRow_memsize, singleRow_memsize, numVertices, hipMemcpyHostToDevice)); //* copy matrix to device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to device: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to device Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);
    //* ---------------------- KERNEL ---------------------- *//
    dim3 dimBlock = dim3(blockSize, blockSize);
    dim3 numBlock = dim3((numVertices + dimBlock.x - 1) / dimBlock.x, (numVertices + dimBlock.y - 1) / dimBlock.y);


    if(vectorize){
        dimBlock = dim3(blockSize >> 2, blockSize);

        cuda(hipEventRecord(start));
        for(int k = 0; k < numVertices; k++)
            FW_simple_kernel_vectorized<<<numBlock, dimBlock>>>((short4*)d_matrix, pitch, numVertices >> 2, k); //* call kernel
    }
    else{
        cuda(hipEventRecord(start));
        for(int k = 0; k < numVertices; k++)
            FW_simple_kernel<<<numBlock, dimBlock>>>(d_matrix, pitch, numVertices, k); //* call kernel
    }

    //* ---------------------------------------------------- *//

    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("Total kernel call: ");
    times.push_back(elapsedTime);
    outputs.push_back("Total kernel call Bandwidth: ");
    if(vectorize)
        times.push_back(8 * numVertices * memsize / elapsedTime / 1.0e6 / 4);
    else
        times.push_back(3 * numVertices * memsize / elapsedTime / 1.0e6);

    cuda(hipHostMalloc(&h_matrix, memsize)); //* allocate memory on host

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(h_matrix, singleRow_memsize, d_matrix, pitch, singleRow_memsize, numVertices, hipMemcpyDeviceToHost)); //* copy matrix to host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to Host: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to Host Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    std::string title =  "Starting SIMPLE FW KERNEL with " + std::to_string(numVertices) +\
    " nodes" + (vectorize ? " with vectorization" : "");
    printMetrics(title, outputs, times); //* print metrics

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}


short* blocked_parallel_FW(const short *g, ll numVertices, int blockSize, bool vectorize){
    size_t pitch = 0;
    short *d_matrix, *h_matrix;
    const size_t singleRow_memsize = numVertices * sizeof(short);
    const size_t memsize = numVertices * numVertices * sizeof(short);


    float elapsedTime;
    std::vector<float> times;
    std::vector<std::string> outputs;

    hipEvent_t start, stop;
    cuda(hipEventCreate(&start));
    cuda(hipEventCreate(&stop));

    cuda(hipMallocPitch(&d_matrix, &pitch, singleRow_memsize, numVertices)); //* allocate memory on device

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(d_matrix, pitch, g, singleRow_memsize, singleRow_memsize, numVertices, hipMemcpyHostToDevice)); //* copy matrix to device
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to device: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to device Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);


    //* ---------------------- KERNEL ---------------------- *//
    const int numBlocks = (numVertices + blockSize - 1) / blockSize;

    dim3 dimBlock = dim3(blockSize, blockSize);
    dim3 dimBlock_phase3 = dim3(numBlocks, numBlocks);
    const size_t sharedMemSize = blockSize * blockSize * sizeof(short);

    if(vectorize){
        dimBlock = dim3(blockSize >> 2, blockSize);

        cuda(hipEventRecord(start));
        for(int k = 0; k < numBlocks; k++){
            blocked_FW_phase1_vectorized<<<1, dimBlock, sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase2_vectorized<<<numBlocks, dimBlock, 2 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase3_vectorized<<<dimBlock_phase3, dimBlock, 3 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
        }
    }
    else{
        cuda(hipEventRecord(start));
        for(int k = 0; k < numBlocks; k++){
            blocked_FW_phase1<<<1, dimBlock, sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase2<<<numBlocks, dimBlock, 2 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
            blocked_FW_phase3<<<dimBlock_phase3, dimBlock, 3 * sharedMemSize>>>(d_matrix, pitch, pitch / sizeof(short), k, blockSize);
        }
    }
    //* ------------------------------------------------------ *//

    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("Total kernel call: ");
    times.push_back(elapsedTime);
    outputs.push_back("Total kernel call Bandwidth: ");

    float bandwidth = 0;
    if(vectorize){
        bandwidth += (2 * numBlocks * memsize) / elapsedTime / 1.0e6 / 4; // Fase 1
        bandwidth += (6 * numBlocks * memsize) / elapsedTime / 1.0e6 / 4; // Fase 2 (3 per 2.1 e 3 per 2.2)
        bandwidth += (4 * numBlocks * memsize) / elapsedTime / 1.0e6 / 4; // Fase 3
    }
    else{
        bandwidth += (2 * numBlocks * memsize) / elapsedTime / 1.0e6; // Fase 1
        bandwidth += (6 * numBlocks * memsize) / elapsedTime / 1.0e6; // Fase 2 (3 per 2.1 e 3 per 2.2)
        bandwidth += (4 * numBlocks * memsize) / elapsedTime / 1.0e6; // Fase 3
    }
    times.push_back(bandwidth);

    cuda(hipHostMalloc(&h_matrix, memsize)); //* allocate memory on host

    cuda(hipEventRecord(start));
    cuda(hipMemcpy2D(h_matrix, singleRow_memsize, d_matrix, pitch, singleRow_memsize, numVertices, hipMemcpyDeviceToHost)); //* copy matrix to host
    cuda(hipEventRecord(stop));
    cuda(hipEventSynchronize(stop));
    cuda(hipEventElapsedTime(&elapsedTime, start, stop));

    outputs.push_back("CudaMemCpy to host: ");
    times.push_back(elapsedTime);
    outputs.push_back("CudaMemCpy to host Bandwidth: ");
    times.push_back(memsize / elapsedTime / 1.0e6);

    std::string title =  "Starting BLOCKED FW KERNEL with " + std::to_string(numVertices) +\
    " nodes" + (vectorize ? " with vectorization" : "");
    printMetrics(title, outputs, times); //* print metrics

    cuda(hipEventDestroy(start));
    cuda(hipEventDestroy(stop));
    cuda(hipFree(d_matrix));
    return h_matrix;
}
